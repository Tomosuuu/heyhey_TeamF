#include "hip/hip_runtime.h"
//参考サイト
/*http://d.hatena.ne.jp/interleave/20091103/1257259065*/

#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <cstdio>
#include <ctime>
#include <stdint.h>

//#define WIDTH 500
#define BLOCK 1

float h_a[WIDTH*WIDTH];
float h_b[WIDTH*WIDTH];
float h_c[WIDTH*WIDTH];

//static inline void
//print_msec(const char * s,clock_t t)
//{
//double msec = (static_cast<double>(t) / CLOCKS_PER_SEC) * 10000;
//printf("%s : %f\n",s,msec);
//}

__global__ void Kernel1(float *A, float *B, float *C)
{
// G P U で の 行 列 乗 算 （ グ ロ ー バ ル メ モ リ の み 使 用 ）
int x=blockIdx.x*blockDim.x + threadIdx.x;
int y=blockIdx.y*blockDim.y + threadIdx.y;

//printf("x:\n",x);
//printf("y:\n",y);

float tmp =0.0;
  for(int k=0; k<WIDTH; k++){
    int row=k+y*WIDTH;
    int col=x+k*WIDTH;
    tmp += A[row]*B[col];
  }
  C[x+y*WIDTH]= tmp;
}

int main(){
  int i;
  srand(0);
  hipSetDevice(0);
  //時間を図るためにt1~t4を宣言
  clock_t t1,t2,t3,t4,t5,t6,t7,t8,t9,t10;
  float *d_a, *d_b, *d_c;

  hipDeviceSynchronize();
  //t9 = clock();

  hipDeviceSynchronize();
  //t1 = clock();
  hipMalloc((void**)&d_a,sizeof(float)*WIDTH*WIDTH);
  hipMalloc((void**)&d_b,sizeof(float)*WIDTH*WIDTH);
  hipMalloc((void**)&d_c,sizeof(float)*WIDTH*WIDTH);
  hipDeviceSynchronize();
  //t2 = clock();

hipMemset(d_c,0,sizeof(float)*WIDTH*WIDTH);

  //printf("a to b\n");
  for(i=0; i<WIDTH*WIDTH; i++){
    h_a[i] = (float)(rand()%10)/1.0f;
    h_b[i] = (float)(rand()%10)/1.0f;
    //printf("%f\n",h_a[i]);
    //printf("%f\n",h_b[i]);
  }

  hipDeviceSynchronize();
  //t3 = clock();
  hipMemcpy(d_a,h_a,sizeof(float)*WIDTH*WIDTH,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,sizeof(float)*WIDTH*WIDTH,hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  //t4 = clock();
  dim3 grid(WIDTH/BLOCK,WIDTH/BLOCK,1);
  dim3 threads(BLOCK,BLOCK,1);
  hipDeviceSynchronize();
  //t5 = clock();
  Kernel1<<< grid, threads >>>(d_a, d_b, d_c);
  hipDeviceSynchronize();
  //t6 = clock();
  hipDeviceSynchronize();
  //t7 = clock();
  hipMemcpy(h_c,d_c,sizeof(float)*WIDTH*WIDTH,hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  //t8 =clock();
  printf(" G P U計算結果 = %f\n",h_c[WIDTH*WIDTH-1]);

  //for(i=0; i<WIDTH*WIDTH; i++){
    //printf("%f\n",h_c[i]);
  //}

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  hipDeviceSynchronize();
  //t10 = clock();

  //GPUで行列a,b,cのデータを転送
  //print_msec("time1:\n",t2-t1);
  //GPUへのデータ転送
  //print_msec("time2:\n",t4-t3);
  //カーネル関数実行
  //print_msec("time3:\n",t6-t5);
  //行列cのデータ転送
  //print_msec("time4:\n",t8-t7);
  //main関数全体
  //print_msec("time5:\n",t10-t9);
}